#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <climits>
#include <vector>
#include <queue>
#include <algorithm>
#include <hip/hip_runtime.h>

#define MAX_CITIES 10000
#define MAX_EDGES 2000000
#define MAX_PATH 1000
#define MAX_DROPS 100

using namespace std;

struct Edge {
    int to;
    int length;
};

// Graph representation
__device__ __host__ inline int idx(int u, int v, int num_cities) {
    return u * num_cities + v;
}

__device__ void dijkstra(int src, int* edge_tos, int* edge_lens, int* edge_next, int* head,
                         int num_cities, int* dist, int* parent) {
    bool visited[MAX_CITIES] = {false};
    for (int i = 0; i < num_cities; ++i) {
        dist[i] = INT_MAX;
        parent[i] = -1;
    }
    dist[src] = 0;

    for (int count = 0; count < num_cities; ++count) {
        int u = -1;
        for (int i = 0; i < num_cities; ++i) {
            if (!visited[i] && (u == -1 || dist[i] < dist[u]))
                u = i;
        }

        if (u == -1 || dist[u] == INT_MAX) break;
        visited[u] = true;

        for (int e = head[u]; e != -1; e = edge_next[e]) {
            int v = edge_tos[e];
            int len = edge_lens[e];
            if (dist[u] + len < dist[v]) {
                dist[v] = dist[u] + len;
                parent[v] = u;
            }
        }
    }
}

__device__ int find_best_shelter(int* dist, int* is_shelter, int* shelter_capacity, int num_cities) {
    int min_dist = INT_MAX;
    int best_shelter = -1;
    for (int s = 0; s < num_cities; ++s) {
        if (is_shelter[s] && dist[s] < min_dist && shelter_capacity[s] > 0) {
            min_dist = dist[s];
            best_shelter = s;
        }
    }
    return best_shelter;
}

__global__ void evacuate_kernel(int num_populated, int* pop_city, int* population_prime, int* population_elder,
                                 int* edge_heads, int* edge_tos, int* edge_lens, int* edge_next, int* head,
                                 int num_cities, int* is_shelter, int* shelter_capacity, int max_dist_elderly,
                                 long long* path_size, long long* paths, long long* num_drops, long long* drops) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= num_populated) return;

    int src = pop_city[i];
    int remaining_prime = population_prime[i];
    int remaining_elder = population_elder[i];

    int drop_idx = 0;
    bool first_path = true;

    while (remaining_prime + remaining_elder > 0) {
        int dist[MAX_CITIES];
        int parent[MAX_CITIES];
        dijkstra(src, edge_tos, edge_lens, edge_next, head, num_cities, dist, parent);

        int best_shelter = find_best_shelter(dist, is_shelter, shelter_capacity, num_cities);
        if (best_shelter == -1) {
            if (remaining_prime + remaining_elder > 0) {
                drops[i * MAX_DROPS * 3 + drop_idx * 3 + 0] = src;
                drops[i * MAX_DROPS * 3 + drop_idx * 3 + 1] = remaining_prime;
                drops[i * MAX_DROPS * 3 + drop_idx * 3 + 2] = remaining_elder;
                drop_idx++;
            }
            break;
        }

        // Reconstruct path
        int temp_path[MAX_PATH];
        int path_len = 0;
        int node = best_shelter;
        while (node != -1 && path_len < MAX_PATH) {
            temp_path[path_len++] = node;
            if (node == src) break;
            node = parent[node];
        }

        if (node != src) break;

        // Reverse path
        for (int j = 0; j < path_len / 2; ++j) {
            int tmp = temp_path[j];
            temp_path[j] = temp_path[path_len - 1 - j];
            temp_path[path_len - 1 - j] = tmp;
        }

        // Check for mid-path elderly drop
        int total_dist = 0;
        int last_city = temp_path[0];
        bool dropped_midway = false;
        int cutoff_path_len = path_len;

        for (int j = 1; j < path_len; ++j) {
            int curr_city = temp_path[j];
            int edge_len = -1;

            // Find edge length
            for (int e = head[last_city]; e != -1; e = edge_next[e]) {
                if (edge_tos[e] == curr_city) {
                    edge_len = edge_lens[e];
                    break;
                }
            }
            if (edge_len == -1) break; // edge not found

            // 🔒 PREVENT exceeding max elderly range
            if (total_dist + edge_len > max_dist_elderly && remaining_elder > 0) {
                // ✅ drop elderly at last_city — BEFORE moving to curr_city
                drops[i * MAX_DROPS * 3 + drop_idx * 3 + 0] = last_city;
                drops[i * MAX_DROPS * 3 + drop_idx * 3 + 1] = 0;
                drops[i * MAX_DROPS * 3 + drop_idx * 3 + 2] = remaining_elder;
                drop_idx++;
                remaining_elder = 0;
                src = last_city; // evac rest from here next loop
                dropped_midway = true;
                cutoff_path_len = j; // up to (but not including) curr_city
                break;
            }

            total_dist += edge_len;
            last_city = curr_city;
        }


        // Append to global path array
        int append_len = dropped_midway ? cutoff_path_len : path_len;
        int start_idx = first_path ? 0 : 1; // skip repeating src
        for (int j = start_idx; j < append_len; ++j) {
            if (path_size[i] < MAX_PATH)
                paths[i * MAX_PATH + path_size[i]++] = temp_path[j];
        }
        first_path = false;

        if (dropped_midway) continue;

        // Try to drop at shelter
        int want = remaining_prime + remaining_elder;
        int* cap_ptr = &shelter_capacity[best_shelter];
        int give = 0, old_val, new_val;

        while (true) {
            old_val = atomicAdd(cap_ptr, 0);
            if (old_val == 0) break;

            give = min(want, old_val);
            new_val = old_val - give;

            int prev = atomicCAS(cap_ptr, old_val, new_val);
            if (prev == old_val) break;
        }

        if (give > 0) {
            int drop_elder = min(give, remaining_elder);
            give -= drop_elder;
            int drop_prime = min(give, remaining_prime);

            remaining_elder -= drop_elder;
            remaining_prime -= drop_prime;

            drops[i * MAX_DROPS * 3 + drop_idx * 3 + 0] = best_shelter;
            drops[i * MAX_DROPS * 3 + drop_idx * 3 + 1] = drop_prime;
            drops[i * MAX_DROPS * 3 + drop_idx * 3 + 2] = drop_elder;
            drop_idx++;

            src = best_shelter;
        } else {
            // Shelter full — try next
            continue;
        }
    }

    num_drops[i] = drop_idx;
    //path_size[i] = paths
}







int main(int argc, char* argv[]) {
    if (argc < 3) {
        cerr << "Usage: " << argv[0] << " <input_file> <output_file>\n";
        return 1;
    }

    ifstream infile(argv[1]);
    ofstream outfile(argv[2]);

    int num_cities, num_roads;
    infile >> num_cities >> num_roads;

    // Graph structure
    int* edge_heads = new int[num_roads * 2];
    int* edge_tos = new int[num_roads * 2];
    int* edge_lens = new int[num_roads * 2];
    int* edge_next = new int[num_roads * 2];
    int* head = new int[num_cities];
    fill(head, head + num_cities, -1);

    int edge_count = 0;
    for (int i = 0; i < num_roads; i++) {
        int u, v, l, c;
        infile >> u >> v >> l >> c;

        edge_tos[edge_count] = v;
        edge_lens[edge_count] = l;
        edge_next[edge_count] = head[u];
        head[u] = edge_count++;
        
        edge_tos[edge_count] = u;
        edge_lens[edge_count] = l;
        edge_next[edge_count] = head[v];
        head[v] = edge_count++;
    }

    int num_shelters;
    infile >> num_shelters;

    int* is_shelter = new int[num_cities]();
    int* shelter_capacity = new int[num_cities]();

    for (int i = 0; i < num_shelters; i++) {
        int city, cap;
        infile >> city >> cap;
        is_shelter[city] = 1;
        shelter_capacity[city] = cap;
    }

    int num_populated;
    infile >> num_populated;

    int* pop_city = new int[num_populated];
    int* population_prime = new int[num_populated];
    int* population_elder = new int[num_populated];

    for (int i = 0; i < num_populated; i++) {
        infile >> pop_city[i] >> population_prime[i] >> population_elder[i];
        std::cout<<pop_city[i]<<population_prime[i]<<population_elder[i]<<std::endl;
    }

    int max_distance_elderly;
    infile >> max_distance_elderly;

    // Device memory
    int *d_pop_city, *d_population_prime, *d_population_elder;
    int *d_edge_heads, *d_edge_tos, *d_edge_lens, *d_edge_next, *d_head;
    int *d_is_shelter, *d_shelter_capacity;

    hipMalloc(&d_pop_city, num_populated * sizeof(int));
    hipMalloc(&d_population_prime, num_populated * sizeof(int));
    hipMalloc(&d_population_elder, num_populated * sizeof(int));
    hipMalloc(&d_edge_heads, edge_count * sizeof(int));
    hipMalloc(&d_edge_tos, edge_count * sizeof(int));
    hipMalloc(&d_edge_lens, edge_count * sizeof(int));
    hipMalloc(&d_edge_next, edge_count * sizeof(int));
    hipMalloc(&d_head, num_cities * sizeof(int));
    hipMalloc(&d_is_shelter, num_cities * sizeof(int));
    hipMalloc(&d_shelter_capacity, num_cities * sizeof(int));

    hipMemcpy(d_pop_city, pop_city, num_populated * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_population_prime, population_prime, num_populated * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_population_elder, population_elder, num_populated * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_edge_heads, edge_heads, edge_count * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_edge_tos, edge_tos, edge_count * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_edge_lens, edge_lens, edge_count * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_edge_next, edge_next, edge_count * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_head, head, num_cities * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_is_shelter, is_shelter, num_cities * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_shelter_capacity, shelter_capacity, num_cities * sizeof(int), hipMemcpyHostToDevice);

    // Output arrays
    long long* path_size = new long long[num_populated];
    long long* paths = new long long[num_populated * MAX_PATH];
    long long* num_drops = new long long[num_populated];
    long long* drops = new long long[num_populated * MAX_DROPS * 3];

    long long *d_path_size, *d_paths, *d_num_drops, *d_drops;
    hipMalloc(&d_path_size, num_populated * sizeof(long long));
    hipMalloc(&d_paths, num_populated * MAX_PATH * sizeof(long long));
    hipMalloc(&d_num_drops, num_populated * sizeof(long long));
    hipMalloc(&d_drops, num_populated * MAX_DROPS * 3 * sizeof(long long));

    // Kernel launch
    int blockSize = 256;
    int gridSize = (num_populated + blockSize - 1) / blockSize;
    evacuate_kernel<<<gridSize, blockSize>>>(num_populated, d_pop_city, d_population_prime, d_population_elder,
                                              d_edge_heads, d_edge_tos, d_edge_lens, d_edge_next, d_head,
                                              num_cities, d_is_shelter, d_shelter_capacity, max_distance_elderly,
                                              d_path_size, d_paths, d_num_drops, d_drops);

    hipMemcpy(path_size, d_path_size, num_populated * sizeof(long long), hipMemcpyDeviceToHost);
    hipMemcpy(paths, d_paths, num_populated * MAX_PATH * sizeof(long long), hipMemcpyDeviceToHost);
    hipMemcpy(num_drops, d_num_drops, num_populated * sizeof(long long), hipMemcpyDeviceToHost);
    hipMemcpy(drops, d_drops, num_populated * MAX_DROPS * 3 * sizeof(long long), hipMemcpyDeviceToHost);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA Kernel Launch Error: " << hipGetErrorString(err) << std::endl;
    }

    // Synchronize and check for post-launch errors
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        std::cerr << "CUDA Post-Kernel Execution Error: " << hipGetErrorString(err) << std::endl;
    }
    Output results
    for (int i = 0; i < num_populated; ++i) {
        for (int j = 0; j < path_size[i]; ++j)
            outfile << paths[i * MAX_PATH + j] << " ";
        outfile << "\n";
    }

    for (int i = 0; i < num_populated; ++i) {
        for (int j = 0; j < num_drops[i]; ++j) {
            outfile << drops[i * MAX_DROPS * 3 + j * 3 + 0] << " "
                    << drops[i * MAX_DROPS * 3 + j * 3 + 1] << " "
                    << drops[i * MAX_DROPS * 3 + j * 3 + 2] << " ";
        }
        outfile << "\n";

    // outfile << "path_sizes = [";
    // for (int i = 0; i < num_populated; i++) outfile << (i ? ", " : "") << path_size[i];
    // outfile << "]\npaths = [";
    // for (int i = 0; i < num_populated; i++) {
    //     outfile << (i ? ", [" : "[");
    //     for (int j = 0; j < path_size[i]; j++) outfile << (j ? ", " : "") << paths[i * MAX_PATH + j];
    //     outfile << "]";
    // }
    // outfile << "]\nnum_drops = [";
    // for (int i = 0; i < num_populated; i++) outfile << (i ? ", " : "") << num_drops[i];
    // outfile << "]\ndrops = [";
    // for (int i = 0; i < num_populated; i++) {
    //     outfile << (i ? ", [" : "[");
    //     for (int j = 0; j < num_drops[i]; j++) {
    //         int base = i * MAX_DROPS * 3 + j * 3;
    //         outfile << (j ? ", " : "") << "(" << drops[base] << ", " << drops[base + 1] << ", " << drops[base + 2] << ")";
    //     }
    //     outfile << "]";
    // }
    // outfile << "]\n";

    return 0;
}
